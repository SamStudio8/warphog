#include "hip/hip_runtime.h"
__global__ void hamming_distance(char* msa, int num_msa, int l, unsigned short* d, int p, int n, unsigned short* idx_map, unsigned short* idy_map) {

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    // Push threadId to account for PAIRS_PER_THREAD (p)
    //threadId += (blockId * (blockDim.x*blockDim.y) * (p-1)) + ((threadIdx.x + (threadIdx.y * blockDim.x)) * (p-1));

    int count = 0;
    while(count < p){

        int idx = idx_map[threadId];
        int idy = idy_map[threadId];

        if(threadId < n){
            //printf("HELLO I AM THE GPU: thread %d, idx %d, idy %d\\n", threadId, idx, idy);

            unsigned int idd = threadId;

            int base_a = 0;
            int base_b = 0;

            if(idx == idy){
                d[idd] = 0;
            }
            else{

                int msa_a = l*idx;
                int msa_b = l*idy;

                //printf("HELLO I AM THE GPU: block %d, thread %d, p %d, idx %d, idy %d, idd %d\\n", blockId, threadIdx.x, count, idx, idy, idd);

                int i = 0;
                int distance = 0;
                while(i < l){
                    base_a = ord_lookup[msa[msa_a + i]];
                    base_b = ord_lookup[msa[msa_b + i]];

                    // No need for if statement, just add the edit distances directly
                    distance += equivalent_lookup[base_a][base_b];

                    //printf("HELLO I AM ON THE GPU: seq_a@%d=%d(%c) seq_b@%d=%d(%c), lookup[%d][%d]=%d: diffs=%d \\n", msa_a+i, msa[msa_a+i], alphabet[msa[msa_a+i]], msa_b+i, msa[msa_b+i], alphabet[msa[msa_b+i]], msa[msa_a+i], msa[msa_b+i], equivalent_lookup[msa[msa_a+i]][msa[msa_b+i]], *d);
                    i++;
                }
                d[idd] = distance;
                //d[idd] += 1;
            }
        }
        count++;
        threadId += 1;
    }
}
