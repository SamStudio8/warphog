#include "hip/hip_runtime.h"
__global__ void hamming_distance(char* msa, unsigned long num_msa, unsigned int l, unsigned short* d, int p, unsigned long n, unsigned int* idx_map, unsigned int* idy_map) {

    unsigned int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned long threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    // Push threadId to account for PAIRS_PER_THREAD (p)
    //threadId += (blockId * (blockDim.x*blockDim.y) * (p-1)) + ((threadIdx.x + (threadIdx.y * blockDim.x)) * (p-1));

    int count = 0;
    while(count < p){

        unsigned long idx = idx_map[threadId];
        unsigned long idy = idy_map[threadId];

        //printf("HELLO I AM THE GPU: thread %d, idx %d, idy %d\n", threadId, idx, idy, n);
        if(threadId < n){
            int base_a = 0;
            int base_b = 0;

            if(idx == idy){
                d[threadId] = 0;
            }
            else{
                // Ballpark, 500k sequences of len 30k will need 150B integers of address space
                unsigned long long msa_a = l*idx;
                unsigned long long msa_b = l*idy;

                //printf("HELLO I AM THE GPU: block %d, thread %d, p %d, idx %d, idy %d, threadId %d\n", blockId, threadIdx.x, count, idx, idy, threadId);

                unsigned int i = 0; // Only covers genomes to 65k but that's OK for tiny viral genomes
                int distance = 0;
                while(i < l){
                    base_a = ord_lookup[msa[msa_a + i]];
                    base_b = ord_lookup[msa[msa_b + i]];

                    // No need for if statement, just add the edit distances directly
                    distance += equivalent_lookup[base_a][base_b];

                    //printf("HELLO I AM ON THE GPU: seq_a@%d=%d(%c) seq_b@%d=%d(%c), lookup[%d][%d]=%d: diffs=%d \n", msa_a+i, msa[msa_a+i], alphabet[msa[msa_a+i]], msa_b+i, msa[msa_b+i], alphabet[msa[msa_b+i]], msa[msa_a+i], msa[msa_b+i], equivalent_lookup[msa[msa_a+i]][msa[msa_b+i]], *d);
                    i++;
                }
                d[threadId] = distance;
            }
        }
        count++;
        threadId += 1;
    }
}
